#include "hip/hip_runtime.h"
//
// Created by root on 23/03/2020.
//

#include "../Matrix.cuh"
#include <iostream>

__global__ void matrixHadamard(double *a, double *b, double *c, int cr, int cc){

    long x = blockIdx.x * blockDim.x + threadIdx.x; // col
    long y = blockIdx.y * blockDim.y + threadIdx.y; // row

    if(x < cc && y < cr){
        c[y * cc + x] = a[y * cc + x] * b[y * cc + x];
    }

}



Matrix Matrix::hadamard(Matrix m){

    if(this->Columns != m.Columns || this->Rows != m.Rows){
        std::cout << "Cannot multiply hadamard. Invalid size";
        exit(-1);
    }

    static double* c;
    c = (double*) calloc(this->Rows*m.Columns,sizeof(double));

    //Define os endereçoes da memória de vídeo
    double *d_a, *d_b, *d_c;

    //Define o tamanho de cada matriz na memória
    int aSize = this->Rows*this->Columns*sizeof(double);
    int bSize = m.Rows*m.Columns*sizeof(double);
    int cSize = this->Rows*m.Columns*sizeof(double);

    //Aloca espaço na memória de vídeo

    hipMalloc((void**)&d_a, aSize);
    hipMalloc((void**)&d_b, bSize);
    hipMalloc((void**)&d_c, cSize);

    //Move as 2 matrizes para a memória de vídeo alocada

    hipMemcpy(d_a, this->Value, aSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, m.Value, bSize, hipMemcpyHostToDevice);

    //Define as dimensões
    dim3 dimBlock(32,32); // 32x32 -> 1024 Threads
    dim3 dimGrid(this->Rows,m.Columns);

    //Efetua a multiplicação
    matrixHadamard<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, this->Rows, m.Columns);
    //Copia o resultado de volta
    hipMemcpy(c, d_c, cSize, hipMemcpyDeviceToHost);

    //Limpa a memória de vídeo
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    //Salva

    return {m.Columns, this->Rows, c};

}


