#include "hip/hip_runtime.h"
//
// Created by root on 23/03/2020.
//

#include "../Matrix.cuh"
#include <iostream>

__global__ void matrixMultiply(double *a, double *b, double *c, int cr, int cc, int ac, int bc){

    long x = blockIdx.x * blockDim.x + threadIdx.x; // col
    long y = blockIdx.y * blockDim.y + threadIdx.y; // row
    double sum = 0;

    if(x < cc && y < cr){

        for(int k = 0; k<ac; k++){
            sum+= a[y*ac+k] * b[k*bc+x];
        }
        c[y * cc + x] = sum;
    }

}



Matrix Matrix::multiply(Matrix m){

    if(this->Columns != m.Rows){
        std::cout << "Cannot multiply matrix. Invalid size";
        exit(-1);
    }

    static double* c;
    c = (double*) calloc(this->Rows*m.Columns,sizeof(double));

    //Define os endereçoes da memória de vídeo
    double *d_a, *d_b, *d_c;

    //Define o tamanho de cada matriz na memória
    int aSize = this->Rows*this->Columns*sizeof(double);
    int bSize = m.Rows*m.Columns*sizeof(double);
    int cSize = this->Rows*m.Columns*sizeof(double);

    //Aloca espaço na memória de vídeo

    hipMalloc((void**)&d_a, aSize);
    hipMalloc((void**)&d_b, bSize);
    hipMalloc((void**)&d_c, cSize);

    //Move as 2 matrizes para a memória de vídeo alocada

    hipMemcpy(d_a, this->Value, aSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, m.Value, bSize, hipMemcpyHostToDevice);

    //Define as dimensões
    dim3 dimBlock(32,32); // 32x32 -> 1024 Threads
    dim3 dimGrid(this->Rows,m.Columns);

    //Efetua a multiplicação
    matrixMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, this->Rows, m.Columns, this->Columns, m.Columns);
    //Copia o resultado de volta
    hipMemcpy(c, d_c, cSize, hipMemcpyDeviceToHost);

    //Limpa a memória de vídeo
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    //Salva

    return {m.Columns, this->Rows, c};

}


