#include "hip/hip_runtime.h"
//
// Created by root on 23/03/2020.
//

#include "../Matrix.cuh"
#include <iostream>

__global__ void matrixAddScalar(double *a, double b, double *c, int cr, int cc){

    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row


    if(x < cc && y < cr){

        c[y * cc + x] = a[y * cc + x]+b;
    }

}



Matrix Matrix::addScalar(double m){

    static double* c;
    c = (double*) calloc(this->Rows*this->Columns,sizeof(double));

    //Define os endereçoes da memória de vídeo
    double *d_a, *d_c;

    //Define o tamanho de cada matriz e escalar na memória
    long aSize = this->Rows*this->Columns*sizeof(double);
    long cSize = this->Rows*this->Columns*sizeof(double);

    //Aloca espaço na memória de vídeo

    hipMalloc((void**)&d_a, aSize);
    hipMalloc((void**)&d_c, cSize);

    //Move a matriz e o escalar para a memória de vídeo alocada

    hipMemcpy(d_a, this->Value, aSize, hipMemcpyHostToDevice);

    //Define as dimensões
    dim3 dimBlock(32,32); // 32x32 -> 1024 Threads
    dim3 dimGrid(this->Rows,this->Columns);

    //Efetua a multiplicação
    matrixAddScalar<<<dimGrid, dimBlock>>>(d_a, m, d_c, this->Rows, this->Columns);
    //Copia o resultado de volta
    hipMemcpy(c, d_c, cSize, hipMemcpyDeviceToHost);

    //Limpa a memória de vídeo
    hipFree(d_a);
    hipFree(d_c);

    //Salva

    return {this->Columns, this->Rows, c};

}


