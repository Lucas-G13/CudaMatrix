#include "hip/hip_runtime.h"
//
// Created by root on 24/03/2020.
//

#include "../Matrix.cuh"

void Matrix::randomFill(){

    static double* c;
    c = (double*) calloc(Columns*Rows,sizeof(double));

    for(int i = 0; i<(Columns*Rows); i++ ){

        c[i] = (((double) rand() / (RAND_MAX)) + 1)*pow(10, -0.5);

    }

    Value = c;

}

void Matrix::zeros(){

    static double* c;
    c = (double*) calloc(Columns*Rows,sizeof(double));

    for(int i = 0; i<(Columns*Rows); i++ ){

        c[i] = 0;

    }

    Value = c;

}

void Matrix::randomFillSmall(){

    static double* c;
    c = (double*) calloc(Columns*Rows,sizeof(double));

    for(int i = 0; i<(Columns*Rows); i++ ){

        c[i] = (((double) rand() / (RAND_MAX)) + 1) /1000;

    }

    Value = c;

}