#include "hip/hip_runtime.h"
//
// Created by root on 24/03/2020.
//

#include "../Matrix.cuh"

void Matrix::randomFill(double seed){

    static double* c;
    c = (double*) calloc(Columns*Rows,sizeof(double));

    for(int i = 0; i<(Columns*Rows); i++ ){

        c[i] = (((double) rand() / (RAND_MAX)))*pow(seed, -0.5);

    }

    Value = c;

}

void Matrix::zeros(){

    static double* c;
    c = (double*) calloc(Columns*Rows,sizeof(double));

    for(int i = 0; i<(Columns*Rows); i++ ){

        c[i] = 0;

    }

    Value = c;

}

void Matrix::randomFillSmall(){

    static double* c;
    c = (double*) calloc(Columns*Rows,sizeof(double));

    for(int i = 0; i<(Columns*Rows); i++ ){

        c[i] = (((double) rand() / (RAND_MAX)) + 1) /1000;

    }

    Value = c;

}