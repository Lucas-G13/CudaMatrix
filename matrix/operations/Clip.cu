#include "hip/hip_runtime.h"
//
// Created by root on 23/03/2020.
//

#include "../Matrix.cuh"
#include <iostream>

__global__ void matrixClip(double *a, double min, double max, double *c, int cr, int cc){

    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row


    if(x < cc && y < cr){

        if(a[y * cc + x] > max){
            c[y * cc + x] = max;
        }else{
            if(a[y * cc + x] < min){
                c[y * cc + x] = min;
            }else{
                c[y * cc + x] = a[y * cc + x];
            }
        }

    }

}



Matrix Matrix::clip(double min, double max){

    static double* c;
    c = (double*) calloc(this->Rows*this->Columns,sizeof(double));

    //Define os endereçoes da memória de vídeo
    double *d_a, *d_c;

    //Define o tamanho de cada matriz e escalar na memória
    long aSize = this->Rows*this->Columns*sizeof(double);
    long cSize = this->Rows*this->Columns*sizeof(double);

    //Aloca espaço na memória de vídeo

    hipMalloc((void**)&d_a, aSize);
    hipMalloc((void**)&d_c, cSize);

    //Move a matriz e o escalar para a memória de vídeo alocada

    hipMemcpy(d_a, this->Value, aSize, hipMemcpyHostToDevice);

    //Define as dimensões
    dim3 dimBlock(32,32); // 32x32 -> 1024 Threads
    dim3 dimGrid(this->Rows,this->Columns);

    //Efetua a multiplicação
    matrixClip<<<dimGrid, dimBlock>>>(d_a, min, max, d_c, this->Rows, this->Columns);
    //Copia o resultado de volta
    hipMemcpy(c, d_c, cSize, hipMemcpyDeviceToHost);

    //Limpa a memória de vídeo
    hipFree(d_a);
    hipFree(d_c);

    //Salva

    return {this->Columns, this->Rows, c};

}


