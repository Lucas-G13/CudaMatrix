#include "hip/hip_runtime.h"
//
// Created by root on 26/03/2020.
//

#include <iostream>
#include "NeuralNetwork.cuh"


NeuralNetwork::NeuralNetwork(int il, int hl, int ol, double lr){

    this->InputLayers = il;
    this->HiddenLayers = hl;
    this->OutputLayers = ol;
    this->LearningRate = lr;

    static Matrix ih = Matrix(il, hl, nullptr);
    ih.randomFill(il);
    this->IHWeight = ih;

    static Matrix ho = Matrix(hl, ol, nullptr);
    ho.randomFill(hl);
    this->HOWeight = ho;

}

Matrix NeuralNetwork::FeedForward(Matrix inputs) {

    Matrix ih = IHWeight.multiply(inputs);
    ih =  ih.sigmoid();

    Matrix ho = HOWeight.multiply(ih);
    ho = ho.sigmoid();

    return ho;

}

double NeuralNetwork::SupervisedTrain(Matrix inputs, Matrix outputs) {



    Matrix ih = IHWeight.multiply(inputs);
    ih =  ih.sigmoid();

    Matrix ho = HOWeight.multiply(ih);
    ho = ho.sigmoid();

    Matrix oe = outputs.sub(ho);
    Matrix he = HOWeight.transpose().multiply(oe);

   Matrix nhow = HOWeight.add(oe.hadamard(ho).hadamard(ho.subScalarInverse(1)).multiply(ih.transpose()).multiplyScalar(LearningRate));

   HOWeight = nhow;

   Matrix nhiw = IHWeight.add(he.hadamard(ih).hadamard(ih.subScalarInverse(1)).multiply(inputs.transpose()).multiplyScalar(LearningRate));
   IHWeight = nhiw;


    return abs(oe.sumAll())+abs(he.sumAll());

}




