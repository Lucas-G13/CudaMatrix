#include "hip/hip_runtime.h"
//
// Created by root on 23/03/2020.
//

#include <iostream>
#include "matrix/Matrix.cu"
#include "neural/NeuralNetwork.cuh"


int main(){


    NeuralNetwork nn = NeuralNetwork(2, 5, 1, 0.1);

    double inpt[] = {0.01,0.99};
    double inpt2[] = {0.99,0.99};
    double inpt3[] = {0.01,0.01};
    double inpt4[] = {0.99,0.01};

    double out1[] = {0.99};
    double out2[] = {0.01};

    Matrix i1 = Matrix(1,2, inpt);
    Matrix i2 = Matrix(1,2, inpt2);
    Matrix i3 = Matrix(1,2, inpt3);
    Matrix i4 = Matrix(1,2, inpt4);

    Matrix o1 = Matrix(1,1, out1);
    Matrix o2 = Matrix(1,1, out2);


    std::cout << "Iniciando treinamento, Neurônios: 2-->5-->1, LR = 0.1\n";

    //Treinano XOR
    long time = clock();
    double err = 0.0;
    while (true){
    err = 0.0;

    for(int x = 0; x<1000; x++) {
        err += nn.SupervisedTrain(i1, o1);
        err += nn.SupervisedTrain(i2, o2);
        err += nn.SupervisedTrain(i3, o2);
        err += nn.SupervisedTrain(i4, o1);

    }


       err = err/1000;
    std::cout<< err << "\n";

    if(err < 1){
        break;
    }

}

    std::cout << "Convergência encontrada, tempo decorrido: " << float((clock()-time))/CLOCKS_PER_SEC << "s\n";



    nn.FeedForward(i1).print();
    nn.FeedForward(i2).print();
    nn.FeedForward(i3).print();
    nn.FeedForward(i4).print();



    return 0;
}